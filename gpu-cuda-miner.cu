#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif
// Implementations of clmemset and memcopy

#if defined(_MSC_VER)
#define ALIGN(x) __declspec(align(x))
#else
#define ALIGN(x) __attribute__((aligned(x)))
#endif

enum blake2b_constant
{
	BLAKE2B_BLOCKBYTES = 128,
	BLAKE2B_OUTBYTES = 64,
	BLAKE2B_KEYBYTES = 64,
	BLAKE2B_SALTBYTES = 16,
	BLAKE2B_PERSONALBYTES = 16
};

#if __CUDA_ARCH__ >= 320
__device__ __forceinline__
uint64_t rotr64(const uint64_t value, const int offset)
{
	uint2 result;
	if(offset < 32)
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
	}
	else
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
	}
	return __double_as_longlong(__hiloint2double(result.y, result.x));
}
#else
__device__ __forceinline__
uint64_t rotr64(const uint64_t x, const int offset)
{
	uint64_t result;
	asm("{\n\t"
		".reg .b64 lhs;\n\t"
		".reg .u32 roff;\n\t"
		"shr.b64 lhs, %1, %2;\n\t"
		"sub.u32 roff, 64, %2;\n\t"
		"shl.b64 %0, %1, roff;\n\t"
		"add.u64 %0, %0, lhs;\n\t"
		"}\n"
		: "=l"(result) : "l"(x), "r"(offset));
	return result;
}
#endif

#define blocksize 256
#define npt 64

__global__ void __launch_bounds__(blocksize, 4) nonceGrind(uint8_t *const __restrict__ headerIn, uint8_t *const __restrict__ hashOut, const uint8_t *const __restrict__ targ, uint8_t *const __restrict__ nonceOut)
{
	uint8_t headerHash8[32];
	int i;

	// Set nonce
	const uint32_t id = (blockDim.x * blockIdx.x + threadIdx.x)*npt;
	uint32_t nonce = 0;

	uint64_t *headerHash64 = (uint64_t*)headerHash8;
	const uint64_t *block = (uint64_t*)headerIn;
	uint64_t h[8] =
	{
		0x6A09E667F2BDC928, 0xbb67ae8584caa73b,
		0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
	};

	uint64_t v[16], v1[16];

	v1[0] = h[0] + h[4] + block[0]; v1[12] = rotr64(0x510E527FADE68281 ^ v1[0], 32); v1[8] = 0x6a09e667f3bcc908 + v1[12]; v1[4] = rotr64(h[4] ^ v1[8], 24);
	v1[0] = v1[0] + v1[4] + block[1]; v1[12] = rotr64(v1[12] ^ v1[0], 16); v1[8] = v1[8] + v1[12]; v1[4] = rotr64(v1[4] ^ v1[8], 63);
	v1[1] = h[1] + h[5] + block[2]; v1[13] = rotr64(0x9b05688c2b3e6c1f ^ v1[1], 32); v1[9] = 0xbb67ae8584caa73b + v1[13]; v1[5] = rotr64(h[5] ^ v1[9], 24);
	v1[1] = v1[1] + v1[5] + block[3]; v1[13] = rotr64(v1[13] ^ v1[1], 16); v1[9] = v1[9] + v1[13]; v1[5] = rotr64(v1[5] ^ v1[9], 63);
	for(i = 0; i < npt; i++)
	{
		*((uint32_t*)(block + 32)) = id+i;
		v[2] = h[2] + h[6] + block[4]; v[14] = rotr64(0xE07C265404BE4294 ^ v[2], 32); v[10] = 0x3c6ef372fe94f82b + v[14]; v[6] = rotr64(h[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + block[5]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = h[3] + h[7] + block[6]; v[15] = rotr64(0x5be0cd19137e2179 ^ v[3], 32); v[11] = 0xa54ff53a5f1d36f1 + v[15]; v[7] = rotr64(h[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[7]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v1[0] + v1[5] + block[8]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v1[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + block[9]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v1[1] + v[6];            v[12] = rotr64(v1[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];            v[13] = rotr64(v1[13] ^ v[2], 32); v[8] = v1[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v1[4];            v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v1[9] + v[14]; v[4] = rotr64(v1[4] ^ v[9], 24);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[4]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[8]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + block[9]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[6]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + block[1]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + block[0]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + block[2]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[7]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + block[5]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + block[3]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + block[8]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];            v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[0]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + block[5]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + block[2]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + block[3]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + block[6]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + block[7]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[1]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + block[9]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + block[4]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + block[7]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + block[9]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[3]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[1]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + block[2]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + block[6]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + block[5]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + block[4]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[0]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + block[8]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + block[9]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + block[0]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[5]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[7]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + block[2]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + block[4]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + block[1]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + block[6]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[8]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + block[3]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + block[1]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[6]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5];            v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + block[0]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + block[8]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[3]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + block[4]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + block[7]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + block[5]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + block[1]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + block[9]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + block[5]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[1]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5];            v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + block[4]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + block[0]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + block[7]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + block[6]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + block[3]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + block[9]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[2]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + block[8]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[7]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5];            v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + block[1]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + block[3]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[9]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + block[5]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + block[0]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + block[4]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + block[8]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[6]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + block[2]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + block[6]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];            v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[9]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + block[3]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + block[0]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[8]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + block[2]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + block[7]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + block[1]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[4]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + block[5]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + block[2]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[8]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[4]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + block[7]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + block[6]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + block[1]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[5]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + block[9]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + block[3]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + block[0]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + block[0]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + block[1]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[2]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[3]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + block[4]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + block[5]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + block[6]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[7]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + block[8]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + block[9]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];            v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];            v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + block[4]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + block[8]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + block[9]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];            v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];            v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + block[6]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + block[1]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];            v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15];
		v[1] = v[1] + v[6] + block[0]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + block[2]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12];
		v[2] = v[2] + v[7];            v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + block[7]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13];
		v[3] = v[3] + v[4] + block[5]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + block[3]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14];

		h[0] = 0x6A09E667F2BDC928 ^ v[0] ^ v[8];
		h[1] = 0xbb67ae8584caa73b ^ v[1] ^ v[9];
		h[2] = 0x3c6ef372fe94f82b ^ v[2] ^ v[10];
		h[3] = 0xa54ff53a5f1d36f1 ^ v[3] ^ v[11];
		if(((uint32_t*)h)[0] == 0)
		{
			nonce = id + i;
			break;
		}
	}
	// Compare header to target
	if(nonce != 0)
	{
#pragma unroll
		for(i = 0; i < 4; i++)
			headerHash64[i] = h[i];
		*((uint64_t*)nonceOut) = ((uint64_t*)headerIn)[4];
#pragma unroll
		for(i = 0; i < 4; i++)
		{
			((uint64_t*)hashOut)[i] = h[i];
		}
	}
}

void nonceGrindcuda(hipStream_t cudastream, int threads, char *blockHeader, char *headerHash, char *targ, char *nonceOut)
{
	nonceGrind << <threads / blocksize / npt, blocksize, 0, cudastream >> >((uint8_t*)blockHeader, (uint8_t*)headerHash, (uint8_t*)targ, (uint8_t*)nonceOut);
}


