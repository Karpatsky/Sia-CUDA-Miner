#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif

#if defined(_MSC_VER)
#define ALIGN(x) __declspec(align(x))
#else
#define ALIGN(x) __attribute__((aligned(x)))
#endif

__device__ __forceinline__
uint64_t __byte_perm_64(const uint64_t source, const uint32_t grab1, const uint32_t grab2)
{
	uint64_t r;
	uint32_t r1;
	uint32_t r2;

	uint32_t i1;
	uint32_t i2;

	asm("mov.b64 {%0, %1}, %2;" : "=r"(i1), "=r"(i2) : "l"(source));
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r1) : "r"(i1), "r"(i2), "r"(grab1));
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r2) : "r"(i1), "r"(i2), "r"(grab2));
	asm("mov.b64 %0, {%1, %2};" : "=l"(r) : "r"(r1), "r"(r2));

	return r;
}


#if __CUDA_ARCH__ >= 520
__device__ __forceinline__
uint64_t rotr64(const uint64_t value, const int offset)
{
	uint2 result;
	if (offset < 32)
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
	}
	else
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
	}
	return __double_as_longlong(__hiloint2double(result.y, result.x));
}
#else
__device__ __forceinline__
uint64_t rotr64(const uint64_t x, const int offset)
{
	uint64_t result;
	asm("{\n\t"
		".reg .b64 lhs;\n\t"
		".reg .u32 roff;\n\t"
		"shr.b64 lhs, %1, %2;\n\t"
		"sub.u32 roff, 64, %2;\n\t"
		"shl.b64 %0, %1, roff;\n\t"
		"add.u64 %0, %0, lhs;\n\t"
		"}\n"
		: "=l"(result) : "l"(x), "r"(offset));
	return result;
}
#endif

__device__ __forceinline__
uint64_t __swap_hilo(const uint64_t source)
{
	uint64_t r;
	uint32_t s1;
	uint32_t s2;

	asm("mov.b64 {%0, %1}, %2;" : "=r"(s1), "=r"(s2) : "l"(source));
	asm("mov.b64 %0, {%1, %2};" : "=l"(r) : "r"(s2), "r"(s1));

	return r;
}

#define blocksize 128
#define npt 256

__global__ void __launch_bounds__(blocksize, 4) nonceGrind(const uint64_t *const __restrict__ headerIn, uint64_t *const __restrict__ hashOut, uint64_t *const __restrict__ nonceOut, const uint64_t *const __restrict__ v1)
{
	uint64_t header[10], h[4], v[16];

	const uint32_t id = (blockDim.x * blockIdx.x + threadIdx.x)*npt;

#pragma unroll
	for (int i = 0; i < 10; i++)
		header[i] = headerIn[i];

	for (int n = id; n < id + npt; n++)
	{
		((uint32_t*)header)[8] = n;
		v[2] = 0x5BF2CD1EF9D6B596u + header[4]; v[14] = __swap_hilo(~0x1f83d9abfb41bd6bu ^ v[2]); v[10] = 0x3c6ef372fe94f82bu + v[14]; v[6] = __byte_perm_64(0x1f83d9abfb41bd6bu ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[5]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = 0x130C253729B586Au + header[6]; v[15] = __swap_hilo(0x5be0cd19137e2179u ^ v[3]); v[11] = 0xa54ff53a5f1d36f1u + v[15]; v[7] = __byte_perm_64(0x5be0cd19137e2179u ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[7]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v1[0] + v1[5] + header[8]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v1[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[9]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v1[1] + v[6];            v[12] = __swap_hilo(v1[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(v1[13] ^ v[2]); v[8] = v1[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v1[4];            v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v1[9] + v[14]; v[4] = __byte_perm_64(v1[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[4]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[8]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[6]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[0]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[2]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[7]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[5]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[3]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[8]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[0]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[5]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[2]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[3]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[6]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[7]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[1]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[9]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[4]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[7]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[9]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[3]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[1]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[2]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[6]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[5]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[4]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[0]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[8]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[9]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[0]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[5]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[7]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[2]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[4]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[1]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[6]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[8]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[3]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[2]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[6]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[0]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[8]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[3]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[4]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[7]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[5]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[1]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[9]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[5]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[1]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[4]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[0]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[7]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[6]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[3]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[9]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[2]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[8]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];            v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[7]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[1]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[3]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[9]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[5]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[0]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[4]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[8]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[6]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[2]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[6]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[9]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[3]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[0]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[8]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[2]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[7]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[1]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[4]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[5]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[2]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[8]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[4]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[7]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[6]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[1]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[5]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[9]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[3]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[0]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[0]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[1]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[2]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[3]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[4]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[5]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[6]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[7]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[8]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[9]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[4]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[8]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[6];	v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1];	v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15];
		v[0] = v[0] + __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[7];
		v[13] = __swap_hilo(v[13] ^ v[2]);
		v[8] = v[8] + v[13];
		v[2] = v[2] + __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107) + header[7];

		h[0] = 0x6A09E667F2BDC928 ^ v[0] ^ (v[8] + __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076));
		if (*((uint32_t*)h) == 0)
		{
			*nonceOut = header[4];

			hashOut[0] = h[0];
			v[1] = v[1] + v[6] + header[0]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12];
			v[1] = v[1] + __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107) + header[2];
			v[3] = v[3] + v[4] + header[5]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14];
			v[3] = v[3] + __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107) + header[3];
			hashOut[1] = 0xbb67ae8584caa73b ^ v[1] ^ (v[9] + __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076));
			hashOut[2] = 0x3c6ef372fe94f82b ^ v[2] ^ (v[10] + __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076));
			hashOut[3] = 0xa54ff53a5f1d36f1 ^ v[3] ^ (v[11] + __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076));
			return;
		}
	}
}

void nonceGrindcuda(hipStream_t cudastream, uint32_t threads, uint64_t *blockHeader, uint64_t *headerHash, uint64_t *nonceOut, uint64_t *vpre)
{
	nonceGrind << <threads / blocksize / npt, blocksize, 0, cudastream >> >(blockHeader, headerHash, nonceOut, vpre);
}
