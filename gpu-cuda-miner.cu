#include <cstdint>
#include <hip/hip_runtime.h>

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif

#if defined(_MSC_VER)
#define ALIGN(x) __declspec(align(x))
#else
#define ALIGN(x) __attribute__((aligned(x)))
#endif

typedef uint32_t u32;
typedef uint64_t  u64;

__device__ __forceinline__
u64 rotr64a(const u64 a, const u32 n)
{
	u32 il;
	u32 ir;

	asm("mov.b64 {%0, %1}, %2;" : "=r"(il), "=r"(ir) : "l"(a));

	u32 tl;
	u32 tr;

	asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(tl) : "r"(il), "r"(ir), "r"(n));
	asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(tr) : "r"(ir), "r"(il), "r"(n));

	u64 r;

	asm("mov.b64 %0, {%1, %2};" : "=l"(r) : "r"(tl), "r"(tr));

	return r;
}

__device__ __forceinline__
u64 rotr64b(const u64 a, const u32 n)
{
	u32 il;
	u32 ir;

	asm("mov.b64 {%0, %1}, %2;" : "=r"(il), "=r"(ir) : "l"(a));

	u32 tl;
	u32 tr;

	asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(tl) : "r"(ir), "r"(il), "r"(n - 32));
	asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(tr) : "r"(il), "r"(ir), "r"(n - 32));

	u64 r;

	asm("mov.b64 %0, {%1, %2};" : "=l"(r) : "r"(tl), "r"(tr));

	return r;
}

__device__ __forceinline__
u64 __byte_perm_64a(const u64 a, const u32 grab1, const u32 grab2)
{
	u64 r;
	u32 r1;
	u32 r2;

	u32 i1;
	u32 i2;

	asm("mov.b64 {%0, %1}, %2;" : "=r"(i1), "=r"(i2) : "l"(a));
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r1) : "r"(i1), "r"(i2), "r"(grab1));
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r2) : "r"(i1), "r"(i2), "r"(grab2));
	asm("mov.b64 %0, {%1, %2};" : "=l"(r) : "r"(r1), "r"(r2));

	return r;
}

#define blocksize 128
#define npt 256

__global__ void __launch_bounds__(blocksize, 4) nonceGrind(const uint64_t *const __restrict__ headerIn, uint64_t *const __restrict__ hashOut, uint64_t *const __restrict__ nonceOut, const uint64_t *const __restrict__ v1)
{
	uint64_t header[10], h[4], v[32];

	const uint32_t id = (blockDim.x * blockIdx.x + threadIdx.x)*npt;

#pragma unroll
	for (int i = 0; i < 10; i++)
		header[i] = headerIn[i]; 

	for (int n = id; n < id + npt; n++)
	{
		((uint32_t*)header)[8] = n;
		v[2] = 0x5BF2CD1EF9D6B596u + header[4]; v[14] = rotr64b(~0x1f83d9abfb41bd6bu ^ v[2], 32); v[10] = 0x3c6ef372fe94f82bu + v[14]; v[6] = __byte_perm_64a(0x1f83d9abfb41bd6bu ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6] + header[5]; v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = 0x130C253729B586Au + header[6]; v[15] = rotr64b(0x5be0cd19137e2179u ^ v[3], 32); v[11] = 0xa54ff53a5f1d36f1u + v[15]; v[7] = __byte_perm_64a(0x5be0cd19137e2179u ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[7]; v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v1[0] + v1[5] + header[8]; v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v1[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5] + header[9]; v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v1[1] + v[6];            v[12] = rotr64b(v1[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64b(v1[13] ^ v[2], 32); v[8] = v1[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v1[4];            v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v1[9] + v[14]; v[4] = __byte_perm_64a(v1[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[4]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[8]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[6]; v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1]; v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[0]; v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6] + header[2]; v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7] + header[7]; v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[5]; v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4] + header[3]; v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4] + header[8]; v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[0]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[5]; v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6] + header[2]; v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[3]; v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6] + header[6]; v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[7]; v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7] + header[1]; v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[9]; v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4] + header[4]; v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[7]; v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4] + header[9]; v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[3]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[1]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[2]; v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5] + header[6]; v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[5]; v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[4]; v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7] + header[0]; v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4] + header[8]; v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[9]; v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4] + header[0]; v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[5]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[7]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[2]; v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6] + header[4]; v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5] + header[1]; v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[6]; v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7] + header[8]; v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[3]; v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[2]; v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[6]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[0]; v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[8]; v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[3]; v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[4]; v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[7]; v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6] + header[5]; v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[1]; v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4] + header[9]; v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4] + header[5]; v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[1]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[4]; v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[0]; v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5] + header[7]; v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[6]; v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6] + header[3]; v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[9]; v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7] + header[2]; v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[8]; v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4];            v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[7]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6] + header[1]; v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[3]; v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[9]; v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[5]; v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5] + header[0]; v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6] + header[4]; v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[8]; v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7] + header[6]; v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[2]; v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[6]; v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[9]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6] + header[3]; v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[0]; v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[8]; v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5] + header[2]; v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6] + header[7]; v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[1]; v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7] + header[4]; v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4] + header[5]; v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4] + header[2]; v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[8]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[4]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[7]; v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6] + header[6]; v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[1]; v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[5]; v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[9]; v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[3]; v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4] + header[0]; v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[0]; v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4] + header[1]; v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[2]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[3]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[4]; v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6] + header[5]; v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[6]; v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[7]; v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[8]; v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[0] = v[0] + v[5] + header[9]; v[15] = __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432); v[10] = v[10] + v[15]; v[5] = rotr64b(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432); v[11] = v[11] + v[12]; v[6] = rotr64b(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64b(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = __byte_perm_64a(v[7] ^ v[8], 0x2107, 0x6543);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432); v[8] = v[8] + v[13]; v[7] = rotr64b(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432); v[9] = v[9] + v[14]; v[4] = rotr64b(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64b(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = __byte_perm_64a(v[4] ^ v[8], 0x2107, 0x6543);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64a(v[12] ^ v[0], 0x1076, 0x5432); v[8] = v[8] + v[12]; v[4] = rotr64b(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[4]; v[13] = rotr64b(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = __byte_perm_64a(v[5] ^ v[9], 0x2107, 0x6543);
		v[1] = v[1] + v[5] + header[8]; v[13] = __byte_perm_64a(v[13] ^ v[1], 0x1076, 0x5432); v[9] = v[9] + v[13]; v[5] = rotr64b(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = rotr64b(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = __byte_perm_64a(v[6] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64a(v[14] ^ v[2], 0x1076, 0x5432); v[10] = v[10] + v[14]; v[6] = rotr64b(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64b(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = __byte_perm_64a(v[7] ^ v[11], 0x2107, 0x6543);
		v[3] = v[3] + v[7] + header[6];	v[15] = __byte_perm_64a(v[15] ^ v[3], 0x1076, 0x5432); v[11] = v[11] + v[15]; v[7] = rotr64b(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1];	v[15] = rotr64b(v[15] ^ v[0], 32); v[10] = v[10] + v[15];
		v[0] = v[0] + __byte_perm_64a(v[5] ^ v[10], 0x2107, 0x6543);
		v[2] = v[2] + v[7];
		v[13] = rotr64b(v[13] ^ v[2], 32);
		v[8] = v[8] + v[13];
		v[2] = v[2] + rotr64a(v[7] ^ v[8], 24) + header[7];

		h[0] = 0x6A09E667F2BDC928 ^ v[0] ^ (v[8] + __byte_perm_64a(v[13] ^ v[2], 0x1076, 0x5432));
		if (*((uint32_t*)h) == 0)
		{
			*nonceOut = header[4];

			hashOut[0] = h[0];
			v[1] = v[1] + v[6] + header[0]; v[12] = rotr64b(v[12] ^ v[1], 32); v[11] = v[11] + v[12];
			v[1] = v[1] + __byte_perm_64a(v[6] ^ v[11], 0x2107, 0x6543) + header[2];
			v[3] = v[3] + v[4] + header[5]; v[14] = rotr64b(v[14] ^ v[3], 32); v[9] = v[9] + v[14];
			v[3] = v[3] + __byte_perm_64a(v[4] ^ v[9], 0x2107, 0x6543) + header[3];
			hashOut[1] = 0xbb67ae8584caa73b ^ v[1] ^ (v[9] + __byte_perm_64a(v[14] ^ v[3], 0x1076, 0x5432));
			hashOut[2] = 0x3c6ef372fe94f82b ^ v[2] ^ (v[10] + __byte_perm_64a(v[15] ^ v[0], 0x1076, 0x5432));
			hashOut[3] = 0xa54ff53a5f1d36f1 ^ v[3] ^ (v[11] + __byte_perm_64a(v[12] ^ v[1], 0x1076, 0x5432));
			return;
		}
	}
}

void nonceGrindcuda(hipStream_t cudastream, uint32_t threads, uint64_t *blockHeader, uint64_t *headerHash, uint64_t *nonceOut, uint64_t *vpre)
{
	nonceGrind <<<threads / blocksize / npt, blocksize, 0, cudastream >>>(blockHeader, headerHash, nonceOut, vpre);
}
